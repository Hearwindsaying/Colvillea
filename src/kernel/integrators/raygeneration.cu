#include "hip/hip_runtime.h"
#ifdef __INTELLISENSE__
#    define __HIPCC__
#endif

#include ""


#include <owl/owl_device.h>

#include <libkernel/base/owldefs.h>
#include <libkernel/base/ray.h>
#include <libkernel/integrators/raygeneration.cuh>

namespace colvillea
{
namespace kernel
{
__host__ __device__ float3 make_float3(vec3f val)
{
    return float3{val.x, val.y, val.z};
}

__global__ void generateCameraRays(SOAProxy<RayWork> rayworkBuff, int nItems)
{
    int jobId = blockIdx.x * blockDim.x + threadIdx.x;
    if (jobId >= nItems)
        return;

    vec2ui pixelPosi{jobId % width, jobId / width};
    const int pixelIndex = jobId;

    const vec2f screen = (vec2f{pixelPosi} + vec2f{.5f, .5f}) / vec2f(width, height);
    
    // temp
    const vec3f lookFrom(-4.f, -3.f, -2.f);
    const vec3f lookAt(0.f, 0.f, 0.f);
    const vec3f lookUp(0.f, 1.f, 0.f);
    const float cosFovy = 0.66f;

    // ----------- compute variable values  ------------------
    vec3f camera_pos = lookFrom;
    vec3f camera_d00 = normalize(lookAt - lookFrom);
    float aspect     = width / float(height);
    vec3f camera_ddu = cosFovy * aspect * normalize(cross(camera_d00, lookUp));
    vec3f camera_ddv = cosFovy * normalize(cross(camera_ddu, camera_d00));
    camera_d00 -= 0.5f * camera_ddu;
    camera_d00 -= 0.5f * camera_ddv;

    Ray ray;
    ray.o = make_float3(camera_pos);
    ray.d = make_float3(normalize(camera_d00 + screen.u * camera_ddu + screen.v * camera_ddv));

    rayworkBuff.setVar(jobId, RayWork{ray, pixelIndex});
}

__global__ void evaluateEscapedRays(SOAProxyQueue<RayEscapedWork>* escapedRayQueue, uint32_t *outputBuffer)
{
    int jobId = blockIdx.x * blockDim.x + threadIdx.x;
    if (jobId >= escapedRayQueue->size())
        return;

    const RayEscapedWork& escapedRayWork = escapedRayQueue->getWorkSOA().getVar(jobId);

    vec2ui pixelPosi{escapedRayWork.pixelIndex % width, escapedRayWork.pixelIndex / width};
    int    pattern = (pixelPosi.x / 8) ^ (pixelPosi.y / 8);

    vec3f color0{.8f, 0.f, 0.f};
    vec3f color1{.8f, .8f, .8f};
    outputBuffer[escapedRayWork.pixelIndex] = owl::make_rgba((pattern & 1) ? color1 : color0);
}

__global__ void evaluateShading(SOAProxyQueue<EvalShadingWork>* evalShadingWorkQueue, uint32_t* outputBuffer)
{
    int jobId = blockIdx.x * blockDim.x + threadIdx.x;
    if (jobId >= evalShadingWorkQueue->size())
        return;

    const EvalShadingWork& evalShadingWork = evalShadingWorkQueue->getWorkSOA().getVar(jobId);

    outputBuffer[evalShadingWork.pixelIndex] =
        owl::make_rgba(
            .2f + .8f * fabs(dot(vec3f(evalShadingWork.rayDirection), vec3f(evalShadingWork.ng))) *
        vec3f(0.0, 0.0, 0.5));
}

} // namespace kernel
} // namespace colvillea